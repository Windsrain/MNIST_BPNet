#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <windows.h>
#include "hip/hip_runtime.h"
#include ""

struct Matrix
{
	int width;
	int height;
	double *elements;
};

__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, double value)
{
	A->elements[row * A->width + col] = value;
}

__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		Cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, Cvalue);
}

int main()
{
 //   hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 128);
	int width = 1 << 10;
	int height = 1 << 10;
	Matrix *A, *B, *C, *D, *E, *F, *G;
	// 申请托管内存
	hipError_t a = hipMallocManaged((void**)&A, sizeof(Matrix));
	hipError_t b = hipMallocManaged((void**)&B, sizeof(Matrix));
	hipError_t c = hipMallocManaged((void**)&C, sizeof(Matrix));
    hipMallocManaged((void**)&D, sizeof(Matrix));
    hipMallocManaged((void**)&E, sizeof(Matrix));
    hipMallocManaged((void**)&F, sizeof(Matrix));
    hipMallocManaged((void**)&G, sizeof(Matrix));
	std::cout << hipGetErrorString(a) << std::endl << hipGetErrorString(b) << std::endl << hipGetErrorString(c) << std::endl;
	int nBytes = width * height * sizeof(double);
	hipError_t d = hipMallocManaged((void**)&A->elements, nBytes);
	hipError_t e = hipMallocManaged((void**)&B->elements, nBytes);
	hipError_t f = hipMallocManaged((void**)&C->elements, nBytes);
	std::cout << hipGetErrorString(d) << std::endl << hipGetErrorString(e) << std::endl << hipGetErrorString(f) << std::endl;
    hipError_t d1 = hipMallocManaged((void**)&D->elements, nBytes);
    std::cout << hipGetErrorString(d1) << std::endl;
    hipError_t e1 = hipMallocManaged((void**)&E->elements, nBytes);
    std::cout << hipGetErrorString(e1) << std::endl;
    hipError_t f1 = hipMallocManaged((void**)&F->elements, nBytes);
    std::cout << hipGetErrorString(f1) << std::endl;
    hipError_t g1 = hipMallocManaged((void**)&G->elements, nBytes);
    std::cout << hipGetErrorString(g1) << std::endl;
	// 初始化数据
	A->height = height;
	A->width = width;
	B->height = height;
	B->width = width;
	C->height = height;
	C->width = width;
	D->height = height;
	D->width = width;
	E->height = height;
	E->width = width;
	F->height = height;
	F->width = width;
	for (int i = 0; i < width * height; ++i)
	{
		A->elements[i] = 1.0;
		B->elements[i] = 2.0;
		D->elements[i] = 1.0;
		E->elements[i] = 2.0;
	}

	// 定义kernel的执行配置
	dim3 blockSize(32, 32);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);
	// 执行kernel
	matMulKernel << < gridSize, blockSize >> > (A, B, C);
	matMulKernel << < gridSize, blockSize >> > (D, E, F);

	// 同步device 保证结果能正确访问
	hipDeviceSynchronize();
	// 检查执行结果
	float maxError = 0.0;
	for (int i = 0; i < width * height; ++i)
		maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
		for (int i = 0; i < width * height; ++i)
		maxError = fmax(maxError, fabs(F->elements[i] - 2 * width));
	
	std::cout << "Error Number: " << maxError << std::endl;

	return 0;
}